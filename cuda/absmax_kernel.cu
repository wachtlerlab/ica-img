#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "absmax_kernel.h"

// kernel functions

__global__ void absmax_kernel (const double *in, double *out)
{
  int tid, gid, off;
  extern __shared__ double data[];

  tid = threadIdx.x;
  gid = (blockIdx.x * blockDim.x * 2) + tid*2;
  off = 1;

  // read data from global memory
  data[tid*2] = in[gid];
  data[tid*2+1] = in[gid+1];

  // b-tree result calculation
  for (off = 1; off < blockDim.x*2; off = off << 1)
    {
      __syncthreads ();

      if (tid < (blockDim.x/off))
        {
          int off_x = (tid * off * 2);
	  int off_y = off_x + off;

          data[off_x] = max (abs (data[off_x]), abs (data[off_y]));
        }
    }

  // write memory back to device memory
  __syncthreads ();
  if (tid == 0)
    out[blockIdx.x] = data[0];

}

#if 0
// utility funtions

int
nearest_power_of_two (int v)
{
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

int
calc_cfg (int s, int *blocksPerGrid, int *threadsPerBlock, size_t *sharedMemSize)
{
  int p2 = nearest_power_of_two (s);
  int block_size;
  int n_blocks;

  block_size = p2 < 1024 ? p2 : 1024;
  n_blocks = s / block_size;

  *threadsPerBlock = block_size / 2;
  *blocksPerGrid = n_blocks;
  *sharedMemSize = block_size * sizeof (double);

  printf ("cfg: [%d: %d * %d] <<<%d, %d, %ld>>>\n", p2, block_size, n_blocks,
	  *blocksPerGrid, *threadsPerBlock, *sharedMemSize);

  return *blocksPerGrid;
}

// main entry point for the abs max function
void
absmax (const double *matrix, int s, double *result)
{
  int blocksPerGrid;
  int threadsPerBlock;
  size_t sharedMemSize;
  int rem;
  size_t p2;
  double *data;
  double *out;
  double *test;

  p2 = nearest_power_of_two (s);
  hipMalloc (&data, p2 * sizeof (double));
  hipMemcpy (data, matrix, s * sizeof (double), hipMemcpyHostToDevice);
  hipMemset (data + s, 0, (p2 - s) * sizeof (double));

  rem = calc_cfg (p2, &blocksPerGrid, &threadsPerBlock, &sharedMemSize);
  hipMalloc (&out, blocksPerGrid * sizeof (double));

  absmax_kernel<<<rem, threadsPerBlock, sharedMemSize>>>(data, out);
  hipDeviceSynchronize ();

  hipHostMalloc (&test, blocksPerGrid * sizeof (double));
  hipMemcpy (test, out, blocksPerGrid * sizeof (double), hipMemcpyDeviceToHost);

  for (int i = 0; i < blocksPerGrid; i++)
    printf ("%f | ", test[i]);

  rem = calc_cfg (rem, &blocksPerGrid, &threadsPerBlock, &sharedMemSize);
  data = out;

  

  do {
    hipDeviceSynchronize ();
    absmax_kernel<<<blocksPerGrid, threadsPerBlock, sharedMemSize>>>(data, out);
    rem = calc_cfg (rem, &blocksPerGrid, &threadsPerBlock, &sharedMemSize);
    printf ("REM %d\n", rem);
  } while (rem > 1);

  hipMemcpy (result, out, sizeof (double), hipMemcpyDeviceToHost);
}

#endif
